#include "hip/hip_runtime.h"
#include <nvfunctional>
#include <ftk/numeric/inverse_linear_interpolation_solver.hh>
#include <ftk/numeric/linear_interpolation.hh>
#include <ftk/numeric/clamp.hh>
#include <ftk/numeric/symmetric_matrix.hh>
#include <ftk/numeric/fixed_point.hh>
#include <ftk/numeric/critical_point_type.hh>
#include <ftk/numeric/critical_point_test.hh>
#include <ftk/filters/xgc_blob_filament_tracker.cuh>
#include "common.cuh"
#include "mx4.cuh"

// what are needed in device memory:
// - field-following interpolants for virtual poloidal planes
// - triangles, edges, and vertex coordinates in 2D mesh
// - current and next timestep of scalar, vector, and jacobian fields

using namespace ftk;

typedef xft_ctx_t ctx_t;

template <typename I, typename F>
__device__
bool check_simplex(
    I current_timestep, 
    I i,
    const F factor,
    const I nphi, const I iphi, const I vphi,
    const I m2n0, const I m2n1, const I m2n2,
    const F m2coords[], // m2 vertex coordinates
    const I m2edges[], // list of m2 edges
    const I m2tris[], // list of m2 triangles
    const F psin_[], // normalized psi
    const ftk::xgc_interpolant_t<I, F>* interpolants, // interpolants
    const F *const scalar[2], // current and next scalar
    const F *const vector[2], // current and next grad
    const F *const jacobian[2], // current and next jacobian
    cp_t & cp) // WIP: critical points
{
  // typedef ftk::fixed_point<> fp_t;

  const I np = nphi * iphi * vphi;
  const I m3n0 = m2n0 * np;

  I verts[3], t[3], p[3];
  F rzpt[3][4], f[3], v[3][2], j[3][2][2], psin[3];

  mx4_get_tri(i, verts, np, m2n0, m2n1, m2n2, m2edges, m2tris);

  for (int k = 0; k < 3; k ++) {
    t[k] = verts[k] / m3n0; // time in m4
    const I v3 = verts[k] % m3n0; // vert in m3
    const I v2 = v3 % m2n0; // vert in m2
    p[k] = v3 / m2n0; // poloidal plane
    psin[k] = psin_[v2];

    mx3_get_coords(v3, rzpt[k], m2n0, m2coords);
    rzpt[k][3] = t[k];
    // const int iv = (t[k] == current_timestep) ? 0 : 1;
    const int iv = t[k]; 
    mx3_interpolate<I, F>(
        v3, nphi, iphi, vphi, 
        m2n0, interpolants,
        scalar[iv], vector[iv], jacobian[iv], 
        f[k], v[k], j[k]);
  }

#if 0
  if (i > 100000000 && i < 100000100)
    printf("i=%d, verts=%d, %d, %d, t=%d, %d, %d, f=%f, %f, %f, p=%d, %d, %d, rzpt=%f, %f, %f, %f, %f, %f, %f, %f, %f, %f, %f, %f\n", 
        i, verts[0], verts[1], verts[2], 
        t[0], t[1], t[2],
        f[0], f[1], f[2],
        p[0], p[1], p[2],
        rzpt[0][0], rzpt[0][1], rzpt[0][2], rzpt[0][3], 
        rzpt[1][0], rzpt[1][1], rzpt[1][2], rzpt[1][3],
        rzpt[2][0], rzpt[2][1], rzpt[2][2], rzpt[2][3]);
  return false;
#endif
  // printf("%f, %f, %f\n", f[0], f[1], f[2]);

  // check if peridocial
  bool b0 = false, b1 = false;
  for (int k = 0; k < 3; k ++) {
    if (p[k] == 0) b0 = true;
    else if (p[k] == np - 1) b1 = true;
  }
  if (b0 && b1) // peridical correction
    for (int k = 0; k < 3; k ++) 
      if (p[k] == 0)
        rzpt[k][2] += np;

  int64_t vf[3][2];
  for (int k = 0; k < 3; k ++) 
    for (int l = 0; l < 2; l ++) 
      vf[k][l] = factor * v[k][l];
 
  // if (v[0][0] > 0)
  //   printf("v=%f, %f, %f, %f, %f, %f\n", 
  //       v[0][0], v[0][1], v[1][0], v[1][1], v[2][0], v[2][1]);

  // if (vf[0][0] + vf[0][1] + vf[1][0] + vf[1][1] + vf[2][0] + vf[2][1] != 0)
  //   printf("vf=%lld, %lld; %lld, %lld; %lld, %lld\n", 
  //       vf[0][0], vf[0][1], vf[1][0], vf[1][1], vf[2][0], vf[2][1]);

  bool succ = ftk::robust_critical_point_in_simplex2(vf, verts);
  if (!succ) return false;

  F mu[3], x[4];
  bool succ2 = ftk::inverse_lerp_s2v2(v, mu);
  ftk::clamp_barycentric<3>(mu);

  ftk::lerp_s2v4(rzpt, mu, x);
  for (int k = 0; k < 3; k ++)
    cp.x[k] = x[k];
  cp.t = x[3];

  // cp.scalar[0] = f[0] * mu[0] + f[1] * mu[1] + f[2] * mu[2];
  cp.scalar[0] = ftk::lerp_s2( f, mu );
  cp.scalar[1] = ftk::lerp_s2( psin, mu );
  cp.tag = i;

  F h[2][2];
  ftk::lerp_s2m2x2(j, mu, h);
  // ftk::make_symmetric2x2(h);
  cp.type = ftk::critical_point_type_2d(h, true);
 
  // printf("tag=%d, verts=%d, %d, %d, x=%f, %f, %f, %f, scalar=%f\n", 
  //     i, verts[0], verts[1], verts[2], cp.x[0], cp.x[1], cp.x[2], cp.x[3], cp.scalar[0]);
  // printf("cp.x=%f, %f, %f, %f, scalar=%f, type=%d\n", 
  //     cp.x[0], cp.x[1], cp.x[2], cp.x[3], cp.scalar[0], 
  //     cp.type);

  return true;
}

template <typename I, typename F>
__global__
void sweep_simplices(
    int scope, 
    I current_timestep, 
    const F factor,
    const I nphi, const I iphi, const I vphi,
    const I m2n0, const I m2n1, const I m2n2,
    const F m2coords[], // m2 vertex coordinates
    const I m2edges[], // list of m2 edges
    const I m2tris[], // list of m2 triangles
    const F psin[],
    const ftk::xgc_interpolant_t<I, F> *interpolants, 
    const F* scalar0, // current scalar
    const F* scalar1, // next scalar
    const F* vector0, 
    const F* vector1, 
    const F* jacobian0, 
    const F* jacobian1,
    unsigned long long &ncps, 
    cp_t *cps)
{
  const I np = nphi * iphi * vphi;
  const I mx3n0 = m2n0 * np; 
  const I mx3n1 = (2 * m2n1 + m2n0) * np; 
  const I mx3n2 = (3 * m2n2 + 2 * m2n1) * np;
  const I mx4n2 = 3 * mx3n2 + 2 * mx3n1;

  int tid = getGlobalIdx_3D_1D();
  I i = tid;
  if (scope == scope_interval) {
    i += mx3n2;
    if (i >= mx4n2) return; // invalid element
  } else { // ordinal
    if (i >= mx3n2) return;
  }
 
  const F* const scalar[2] = {scalar0, scalar1};
  const F* const vector[2] = {vector0, vector1};
  const F* const jacobian[2] = {jacobian0, jacobian1};

  cp_t cp;
  bool succ = check_simplex<I, F>(
      current_timestep, 
      i, 
      factor,
      nphi, iphi, vphi, 
      m2n0, m2n1, m2n2, 
      m2coords, m2edges, m2tris,
      psin,
      interpolants, 
      scalar, vector, jacobian, 
      cp);
  
  if (succ) {
    unsigned long long idx = atomicAdd(&ncps, 1ul);
    cp.tag = i; // tid;
    cps[idx] = cp;
  }
}

void xft_create_ctx(ctx_t **c_, int device, int device_buffer_size_in_mb)
{
  *c_ = (ctx_t*)malloc(sizeof(ctx_t));
  ctx_t *c = *c_;

  c->device = device;
  hipSetDevice(device);

  hipMalloc((void**)&c->dncps, sizeof(unsigned long long));
  hipMemset(c->dncps, 0, sizeof(unsigned long long));
  checkLastCudaError("[FTK-CUDA] cuda malloc");

  c->bufsize = device_buffer_size_in_mb * size_t(1024 * 1024); 
  c->hcps = (cp_t*)malloc(c->bufsize);
  hipMalloc((void**)&c->dcps, c->bufsize);
  checkLastCudaError("[FTK-CUDA] cuda malloc: creating buffer");

  c->d_psin = NULL;

  c->d_kernel_nodes = NULL;
  c->d_kernel_values = NULL;
  c->d_kernel_lengths = NULL;
  c->d_kernel_offsets = NULL;

  c->d_scalar_in = NULL;
  c->d_scalar[0] = NULL;
  c->d_scalar[1] = NULL;
  c->d_vector[0] = NULL;
  c->d_vector[1] = NULL;
  c->d_jacobian[0] = NULL;
  c->d_jacobian[1] = NULL;

  c->factor = 32768.0;
}

void xft_destroy_ctx(ctx_t **c_)
{
  ctx_t *c = *c_;

  if (c->d_m2coords != NULL) hipFree(c->d_m2coords);
  if (c->d_m2edges != NULL) hipFree(c->d_m2edges);
  if (c->d_m2tris != NULL) hipFree(c->d_m2tris);
  if (c->d_psin != NULL) hipFree(c->d_psin);

  if (c->d_interpolants != NULL) hipFree(c->d_interpolants);

  if (c->d_kernel_nodes != NULL) hipFree(c->d_kernel_nodes);
  if (c->d_kernel_values != NULL) hipFree(c->d_kernel_values);
  if (c->d_kernel_lengths != NULL) hipFree(c->d_kernel_lengths);
  if (c->d_kernel_offsets != NULL) hipFree(c->d_kernel_offsets);

  if (c->d_scalar_in != NULL) hipFree(c->d_scalar_in);
  if (c->d_scalar[0] != NULL) hipFree(c->d_scalar[0]);
  if (c->d_scalar[1] != NULL) hipFree(c->d_scalar[1]);
  if (c->d_vector[0] != NULL) hipFree(c->d_vector[0]);
  if (c->d_vector[1] != NULL) hipFree(c->d_vector[1]);
  if (c->d_jacobian[0] != NULL) hipFree(c->d_jacobian[0]);
  if (c->d_jacobian[1] != NULL) hipFree(c->d_jacobian[1]);
  
  checkLastCudaError("[FTK-CUDA] cuda free");

  free(*c_);
  *c_ = NULL;
}

void xft_execute(ctx_t *c, int scope, int current_timestep)
{
  const int np = c->nphi * c->iphi * c->vphi;
  const int mx3n1 = (2 * c->m2n1 + c->m2n0) * np;
  const int mx3n2 = (3 * c->m2n2 + 2 * c->m2n1) * np;
  // const int mx4n2 = 3 * mx3n2 + 2 * mx3n1;
  const int mx4n2_ordinal  = mx3n2, 
            mx4n2_interval = 2 * mx3n2 + 2 * mx3n1;
  // fprintf(stderr, "executing timestep %d\n", current_timestep);

  size_t ntasks;
  if (scope == scope_ordinal) ntasks = mx4n2_ordinal;
  else ntasks = mx4n2_interval;
  
  fprintf(stderr, "ntasks=%zu\n", ntasks);
  
  const int maxGridDim = 1024;
  const int blockSize = 256;
  const int nBlocks = idivup(ntasks, blockSize);
  dim3 gridSize;

  if (nBlocks >= maxGridDim) gridSize = dim3(idivup(nBlocks, maxGridDim), maxGridDim);
  else gridSize = dim3(nBlocks);

  sweep_simplices<int, double><<<gridSize, blockSize>>>(
      scope, current_timestep, 
      c->factor,
      c->nphi, c->iphi, c->vphi, 
      c->m2n0, c->m2n1, c->m2n2, 
      c->d_m2coords, c->d_m2edges, c->d_m2tris, 
      c->d_psin,
      c->d_interpolants, 
      c->d_scalar[0], c->d_scalar[1],
      c->d_vector[0], c->d_vector[1],
      c->d_jacobian[0], c->d_jacobian[1], 
      *c->dncps, c->dcps);
  hipDeviceSynchronize();
  checkLastCudaError("[FTK-CUDA] sweep_simplicies");

  hipMemcpy(&c->hncps, c->dncps, sizeof(unsigned long long), hipMemcpyDeviceToHost);
  hipMemset(c->dncps, 0, sizeof(unsigned long long)); // clear the counter
  checkLastCudaError("[FTK-CUDA] cuda memcpy device to host, 1");
  fprintf(stderr, "ncps=%llu\n", c->hncps);
  hipMemcpy(c->hcps, c->dcps, sizeof(cp_t) * c->hncps, hipMemcpyDeviceToHost);
  
  checkLastCudaError("[FTK-CUDA] cuda memcpy device to host, 2");
}

void xft_swap(ctx_t *c)
{
  // fprintf(stderr, "calling swap\n");
  std::swap(c->d_scalar[0], c->d_scalar[1]);
  std::swap(c->d_vector[0], c->d_vector[1]);
  std::swap(c->d_jacobian[0], c->d_jacobian[1]);
}

void xft_load_data(ctx_t *c, 
    const double *scalar, const double *vector, const double *jacobian)
{
  double *dd_scalar;
  if (c->d_scalar[0] == NULL) {
    hipMalloc((void**)&c->d_scalar[0], sizeof(double) * size_t(c->m2n0) * size_t(c->nphi));
    checkLastCudaError("[FTK-CUDA] loading scalar field data, malloc 0");
    dd_scalar = c->d_scalar[0];
  } else if (c->d_scalar[1] == NULL) {
    hipMalloc((void**)&c->d_scalar[1], sizeof(double) * size_t(c->m2n0) * size_t(c->nphi));
    checkLastCudaError("[FTK-CUDA] loading scalar field data, malloc 0.1");
    dd_scalar = c->d_scalar[1];
  } else {
    std::swap(c->d_scalar[0], c->d_scalar[1]);
    dd_scalar = c->d_scalar[1];
  }
  // fprintf(stderr, "dd=%p, d0=%p, d1=%p, src=%p\n", dd_scalar, c->d_scalar[0], c->d_scalar[1], scalar);
  hipMemcpy(dd_scalar, scalar, sizeof(double) * size_t(c->m2n0 * c->nphi), 
      hipMemcpyHostToDevice);
  checkLastCudaError("[FTK-CUDA] loading scalar field data, memcpy 0");
 
  /// 
  double *dd_vector;
  if (c->d_vector[0] == NULL) {
    hipMalloc((void**)&c->d_vector[0], sizeof(double) * size_t(c->m2n0) * size_t(c->nphi) * 2);
    dd_vector = c->d_vector[0];
  } else if (c->d_vector[1] == NULL) {
    hipMalloc((void**)&c->d_vector[1], sizeof(double) * size_t(c->m2n0) * size_t(c->nphi)* 2);
    dd_vector = c->d_vector[1];
  } else {
    std::swap(c->d_vector[0], c->d_vector[1]);
    dd_vector = c->d_vector[1];
  }
  hipMemcpy(dd_vector, vector, sizeof(double) * size_t(c->m2n0 * c->nphi * 2), 
      hipMemcpyHostToDevice);
  checkLastCudaError("[FTK-CUDA] loading vector field data");

  /// 
  double *dd_jacobian;
  if (c->d_jacobian[0] == NULL) {
    hipMalloc((void**)&c->d_jacobian[0], sizeof(double) * size_t(c->m2n0) * size_t(c->nphi) * 4);
    dd_jacobian = c->d_jacobian[0];
  } else if (c->d_jacobian[1] == NULL) {
    hipMalloc((void**)&c->d_jacobian[1], sizeof(double) * size_t(c->m2n0) * size_t(c->nphi) * 4);
    dd_jacobian = c->d_jacobian[1];
  } else {
    std::swap(c->d_jacobian[0], c->d_jacobian[1]);
    dd_jacobian = c->d_jacobian[1];
  }
  hipMemcpy(dd_jacobian, jacobian, sizeof(double) * size_t(c->m2n0 * c->nphi) * 4, 
      hipMemcpyHostToDevice);
  checkLastCudaError("[FTK-CUDA] loading jacobian field data");
}

template <typename I, typename F>
__global__
void smooth_scalar_vector_jacobian(
    const I nphi,
    const I m2n0,
    const F *m2coords,
    const F sigma,
    const size_t *lengths,
    const size_t *offsets,
    const I *nodes,
    const F *values,
    const F *scalar_in,
    F *scalar_out,
    F *vector_out, 
    F *jacobian_out)
{
  int idx = getGlobalIdx_3D_1D();
  if (idx >= m2n0 * nphi) return; // out of bounds

  const F sigma2 = sigma * sigma, 
          sigma4 = sigma2 * sigma2;

  const I i = idx % m2n0, 
          p = idx / m2n0;

  // scalar_out[idx] = 0; // assuming out values are zero'ed
  for (int j = 0; j < lengths[i]; j ++) {
    const int k = nodes[offsets[i] + j];
    const F w = values[offsets[i] + j]; // weight
    const F d[2] = {m2coords[k*2] - m2coords[i*2], m2coords[k*2+1] - m2coords[i*2+1]};
  
    const F f = scalar_in[ k + p * m2n0 ];
    scalar_out[idx] += f * w;
    
    // if (idx == 10000)
    // if (scalar_in[idx] > 0.01)
    //   printf("w[%d]=%f, scalar_in=%f\n", k, w, scalar_in[idx]);

    vector_out[idx*2]   += -f * w * d[0] / sigma2;
    vector_out[idx*2+1] += -f * w * d[1] / sigma2;

    jacobian_out[idx*4]   += (d[0]*d[0] / sigma2 - 1) / sigma2 * f * w;
    jacobian_out[idx*4+1] += d[0]*d[1] / sigma4 * f * w;
    jacobian_out[idx*4+2] += d[0]*d[1] / sigma4 * f * w;
    jacobian_out[idx*4+3] += (d[1]*d[1] / sigma2 - 1) / sigma2 * f * w;
  }
    
  // if (scalar_out[idx] > 0.01)
  //   printf("%f, %f, %f\n", scalar_out[idx], vector_out[idx*2], vector_out[idx*2+1]);
}

void xft_smooth_scalar_vector_jacobian(ctx_t *c, 
    const double *d_scalar_in, 
    double *d_scalar_out, 
    double *d_vector_out, 
    double *d_jacobian_out)
{
  hipMemset(d_scalar_out, 0, size_t(c->m2n0 * c->nphi) * sizeof(double));
  hipMemset(d_vector_out, 0, size_t(2 * c->m2n0 * c->nphi) * sizeof(double));
  hipMemset(d_jacobian_out, 0, size_t(4 * c->m2n0 * c->nphi) * sizeof(double));
  
  size_t ntasks = c->nphi * c->m2n0;
  const int maxGridDim = 1024;
  const int blockSize = 256;
  const int nBlocks = idivup(ntasks, blockSize);
  dim3 gridSize;
  if (nBlocks >= maxGridDim) gridSize = dim3(idivup(nBlocks, maxGridDim), maxGridDim);
  else gridSize = dim3(nBlocks);

  smooth_scalar_vector_jacobian<int, double><<<gridSize, blockSize>>>(
      c->nphi, 
      c->m2n0, 
      c->d_m2coords,
      c->sigma,
      c->d_kernel_lengths,
      c->d_kernel_offsets,
      c->d_kernel_nodes,
      c->d_kernel_values,
      d_scalar_in, 
      d_scalar_out,
      d_vector_out,
      d_jacobian_out);
  hipDeviceSynchronize();
  checkLastCudaError("[FTK-CUDA] smoothing scalar vector jacobian");

  // update scaling factor; CPU implementation for now..
  double *h_vector = (double*)malloc(size_t(c->m2n0 * c->nphi) * sizeof(double));
  hipMemcpy(h_vector, d_vector_out, size_t(c->m2n0 * c->nphi) * sizeof(double), 
      hipMemcpyDeviceToHost);
  double maxabs = 0.0;
  for (int i = 0; i < c->m2n0 * c->nphi; i ++)
    maxabs = std::max(maxabs, std::abs(h_vector[i]));
  free(h_vector);
  
  double factor = std::exp2(-std::ceil(std::log2(maxabs)) + 20); // 20 bits
  c->factor = std::max(c->factor, factor);

  std::cerr << "maxabs: " << maxabs << ", factor: " << c->factor << std::endl;
}

void xft_load_smoothing_kernel(ctx_t *c, double sigma, const std::vector<std::vector<std::tuple<int, double>>>& kernels)
{
  c->sigma = sigma;

  // fprintf(stderr, "loading smoothing kernels to GPU...\n");
  std::vector<size_t> lengths(kernels.size()), offsets(kernels.size());
  std::vector<int> nodes;
  std::vector<double> values;

  size_t acc = 0;
  for (size_t i = 0; i < kernels.size(); i ++) { // nodes
    const std::vector<std::tuple<int, double>>& kernel = kernels[i];
    lengths[i] = kernel.size();
    offsets[i] = acc;
    acc += kernel.size();

    for (size_t j = 0; j < kernel.size(); j ++) {
      nodes.push_back(std::get<0>(kernel[j]));
      values.push_back(std::get<1>(kernel[j]));
    }
  }
  
  hipMalloc((void**)&c->d_kernel_nodes, nodes.size() * sizeof(int));
  hipMemcpy(c->d_kernel_nodes, nodes.data(), nodes.size() * sizeof(int), hipMemcpyHostToDevice);

  hipMalloc((void**)&c->d_kernel_values, values.size() * sizeof(double));
  hipMemcpy(c->d_kernel_values, values.data(), values.size() * sizeof(double), hipMemcpyHostToDevice);

  hipMalloc((void**)&c->d_kernel_lengths, lengths.size() * sizeof(size_t));
  hipMemcpy(c->d_kernel_lengths, lengths.data(), lengths.size() * sizeof(size_t), hipMemcpyHostToDevice);

  hipMalloc((void**)&c->d_kernel_offsets, offsets.size() * sizeof(size_t));
  hipMemcpy(c->d_kernel_offsets, offsets.data(), offsets.size() * sizeof(size_t), hipMemcpyHostToDevice);

  checkLastCudaError("[FTK-CUDA] loading smoothing kernel");
  // fprintf(stderr, "smoothing kernels loaded to GPU.\n");
}

void xft_load_psin(ctx_t *c, const double *psin)
{
  hipMalloc((void**)&c->d_psin, size_t(c->m2n0) * sizeof(double));
  hipMemcpy(c->d_psin, psin, size_t(c->m2n0) * sizeof(double), hipMemcpyHostToDevice);
  checkLastCudaError("[FTK-CUDA] loading psin");
}

void xft_load_mesh(ctx_t *c,
    int nphi, int iphi, int vphi,
    int m2n0, int m2n1, int m2n2,
    const double *m2coords, const int *m2edges, const int *m2tris)
{
  c->nphi = nphi; 
  c->iphi = iphi;
  c->vphi = vphi;
  c->m2n0 = m2n0;
  c->m2n1 = m2n1;
  c->m2n2 = m2n2;

  hipMalloc((void**)&c->d_m2coords, size_t(m2n0) * sizeof(double) * 2);
  checkLastCudaError("[FTK-CUDA] loading xgc mesh, malloc 0");
  hipMemcpy(c->d_m2coords, m2coords, size_t(m2n0) * sizeof(double) * 2, hipMemcpyHostToDevice);
  checkLastCudaError("[FTK-CUDA] loading xgc mesh, memcpy 0");

  hipMalloc((void**)&c->d_m2edges, size_t(m2n1) * sizeof(int) * 2);
  hipMemcpy(c->d_m2edges, m2edges, size_t(m2n1) * sizeof(int) * 2, hipMemcpyHostToDevice);

  hipMalloc((void**)&c->d_m2tris, size_t(m2n2) * sizeof(int) * 3);
  hipMemcpy(c->d_m2tris, m2tris, size_t(m2n2) * sizeof(int) * 3, hipMemcpyHostToDevice);
  
  checkLastCudaError("[FTK-CUDA] loading xgc mesh");
}

void xft_load_interpolants(ctx_t *c, const std::vector<std::vector<ftk::xgc_interpolant_t<>>> &interpolants)
{
  assert(c->vphi == interpolants.size());

  hipMalloc((void**)&c->d_interpolants, 
      size_t(c->m2n0) * sizeof(ftk::xgc_interpolant_t<>) * c->vphi);
  checkLastCudaError("[FTK-CUDA] loading xgc interpolants, malloc 0");

  for (size_t i = 1; i < interpolants.size(); i ++) {
    hipMemcpy(c->d_interpolants + (i-1) * size_t(c->m2n0), // * sizeof(ftk::xgc_interpolant_t<>), 
        interpolants[i].data(), size_t(c->m2n0) * sizeof(ftk::xgc_interpolant_t<>), hipMemcpyHostToDevice);
    checkLastCudaError("[FTK-CUDA] loading xgc interpolants, memcpy");
  }
  
  checkLastCudaError("[FTK-CUDA] loading xgc interpolants");
}

void xft_load_scalar_data(ctx_t *c, const double *scalar)
{
  // fprintf(stderr, "loading and smoothing scalar data w/ gpu...\n");
  if (c->d_scalar_in == NULL)
    hipMalloc((void**)&c->d_scalar_in, sizeof(double) * size_t(c->m2n0 * c->nphi));
  hipMemcpy(c->d_scalar_in, scalar, sizeof(double) * size_t(c->m2n0 * c->nphi), 
      hipMemcpyHostToDevice);

  double *dd_scalar, *dd_vector, *dd_jacobian;
  if (c->d_scalar[0] == NULL) {
    // fprintf(stderr, "init slot 0\n");
    hipMalloc((void**)&c->d_scalar[0], sizeof(double) * size_t(c->m2n0 * c->nphi));
    hipMalloc((void**)&c->d_vector[0], sizeof(double) * size_t(c->m2n0 * c->nphi * 2));
    hipMalloc((void**)&c->d_jacobian[0], sizeof(double) * size_t(c->m2n0 * c->nphi * 4));
    dd_scalar = c->d_scalar[0];
    dd_vector = c->d_vector[0];
    dd_jacobian = c->d_jacobian[0];
  } else if (c->d_scalar[1] == NULL) {
    // fprintf(stderr, "init slot 1\n");
    hipMalloc((void**)&c->d_scalar[1], sizeof(double) * size_t(c->m2n0 * c->nphi));
    hipMalloc((void**)&c->d_vector[1], sizeof(double) * size_t(c->m2n0 * c->nphi * 2));
    hipMalloc((void**)&c->d_jacobian[1], sizeof(double) * size_t(c->m2n0 * c->nphi * 4));
    dd_scalar = c->d_scalar[1];
    dd_vector = c->d_vector[1];
    dd_jacobian = c->d_jacobian[1];
  } else {
    // fprintf(stderr, "swapping 0 and 1\n");
    std::swap(c->d_scalar[0], c->d_scalar[1]);
    std::swap(c->d_vector[0], c->d_vector[1]);
    std::swap(c->d_jacobian[0], c->d_jacobian[1]);
    dd_scalar = c->d_scalar[1];
    dd_vector = c->d_vector[1];
    dd_jacobian = c->d_jacobian[1];
  }

  xft_smooth_scalar_vector_jacobian(c, 
      c->d_scalar_in, dd_scalar, dd_vector, dd_jacobian);
  
  // fprintf(stderr, "scalar smoothed and loaded to gpu\n");
}
